
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

using namespace std;

typedef unsigned int uint;
typedef unsigned long long ull;

const int PADDING = 16;

__device__ uint LR_dev(uint a, int x){
	return a << x | a >> 32-x;
}

__global__ void sha1_kernel(ull* res, ull IDX, uint h0, uint h1, uint h2, uint h3, uint h4) {
	ull id = threadIdx.x | (ull)blockIdx.x << 10 | (ull)blockIdx.y << 20 | (ull)IDX << 32, idx = id;

	uint w[16];
	for(int i = 0; i < PADDING/4; i++, idx >>= 16){
		w[i] = 0x40404040u | (idx&15) << 24 | (idx>>4&15) << 16 | (idx>>8&15) << 8 | (idx>>12&15);
	}
	w[PADDING/4] = 0x0a800000;
	for(int i = PADDING/4; i < 15; i++) w[i] = 0;
	w[15] = 1800;

	uint a, b, c, d, e, f, k;
	a = h0; b = h1; c = h2; d = h3; e = h4;
	for(int i = 0; i < 16; i++){
		f = (b&c)|(~b&d);
		k = 0x5A827999;
		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	for(int i = 16; i < 20; i++){
		w[i&15] = LR_dev(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b&c)|(~b&d);
		k = 0x5A827999;

		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	for(int i = 20; i < 40; i++){
		w[i&15] = LR_dev(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b^c^d);
		k = 0x6ED9EBA1; 

		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	for(int i = 40; i < 60; i++){
		w[i&15] = LR_dev(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b&c)|(b&d)|(c&d);
		k = 0x8F1BBCDC;

		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	for(int i = 60; i < 80; i++){
		w[i&15] = LR_dev(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = b^c^d;
		k = 0xCA62C1D6;

		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	h0 = h0 + a;
	h1 = h1 + b;
	h2 = h2 + c;
	h3 = h3 + d;
	h4 = h4 + e;
	if(h0 == 0 && (h1 >> 31) == 0) *res = id;
}

struct sha1{
	sha1(uint h0, uint h1, uint h2, uint h3, uint h4){}
	uint h0, h1, h2, h3, h4;
};

string modify_message()
{
	string log = "";
	char c;
	while(scanf("%c", &c) != EOF) log += c;
	while(log.size() % 64 == 0) log += " ";
	for(int t = 0; t < PADDING; t++) log += "@";
	log += '\0';

	int m = 1, l = log.size(), p;
	while(log.size() >= m*10) m *= 10;
	for(p = 7; m; p++, m /= 10) log[p] = l/m + '0', l %= m;
	cout << log << std::endl;

	if(log[p] != '\0'){
		printf("message length's digit is changed while modifying\n");
		printf("Please add or remove +- 50? letters \n");
		exit(-1);
	}
	cout << log << "\n";
	return log;
}

union block{
	uint v[16];
	char s[64];
};

uint h0, h1, h2, h3, h4;

uint LR(uint a, int x){
	return a << x | a >> 32-x;
}

sha1 get_hash(string log){
	block buf[10000] = {};
	h0 = 0x67452301;
	h1 = 0xEFCDAB89;
	h2 = 0x98BADCFE;
	h3 = 0x10325476;
	h4 = 0xC3D2E1F0;

	memcpy(buf, log.c_str(), log.size()*8);
	int len = log.size(), nbits = len * 8;
	buf[0].s[len++] = 0x80;

	uint nblock = (len+7)/64 + 1;
	buf[nblock-1].v[14] = 0;
	buf[nblock-1].v[15] = nbits;

	for(int t = 0; t+1 < nblock; t++){
		block cur = buf[t];
		for(int i = 0; i < 16; i++){
			if(t == nblock-1 && i == 15) continue;
			swap(cur.s[i*4+0], cur.s[i*4+3]);
			swap(cur.s[i*4+1], cur.s[i*4+2]);
		}
		uint w[16];
		for(int i = 0; i < 16; i++) w[i] = cur.v[i];

		uint a, b, c, d, e, f, k;
		a = h0; b = h1; c = h2; d = h3; e = h4;
		for(int i = 0; i < 80; i++){
			if(i >= 16) w[i%16] = LR(w[(i-3+16)%16]^w[(i-8+16)%16]^w[(i-14+16)%16]^w[(i-16+16)%16], 1);
			if(i <= 19){
				f = (b&c)|(~b&d);
				k = 0x5A827999;
			}
			else if(i <= 39){
				f = (b^c^d);
				k = 0x6ED9EBA1; 
			}
			else if(i <= 59){
				f = (b&c)|(b&d)|(c&d);
				k = 0x8F1BBCDC;
			}
			else if(i <= 79){
				f = b^c^d;
				k = 0xCA62C1D6;
			}
			uint tmp = LR(a, 5) + f + e + k + w[i%16];
			e = d; d = c; c = LR(b, 30); b = a; a = tmp;
		}
		h0 = h0 + a;
		h1 = h1 + b;
		h2 = h2 + c;
		h3 = h3 + d;
		h4 = h4 + e;
	}
	return sha1(h0, h1, h2, h3, h4);
}

void run_on_gpu(string log, sha1 hash){
	ull *res;
	ull res_copy;
	dim3 threadsPerBlock(256, 1);
	dim3 numBlocks(4096, 4096);

	hipMalloc(&res, sizeof(ull));

	for(int i = 0;; i++){
		sha1_kernel<<<numBlocks, threadsPerBlock>>>(res, i, hash.h0, hash.h1, hash.h2, hash.h3, hash.h4);

		hipMemcpy(&res_copy, res, sizeof(ull), hipMemcpyDeviceToHost);
		if(res_copy) break;
	}
	cout << log.substr(0, (int)log.size() - PADDING);
	for(int i = 0; i < PADDING; i++){
		printf("%c", 64 | res_copy&15);
		res_copy /= 16;
	}
}

int main()
{
	string log = modify_message();
	sha1 hash = get_hash(log);
	run_on_gpu(log, hash);
}
