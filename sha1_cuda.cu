
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

using namespace std;

typedef unsigned int uint;
typedef unsigned long long ull;

const int PADDING = 16;

__device__ uint LR_dev(uint a, int x){
	return a << x | a >> 32-x;
}

__global__ void sha1_kernel(ull* res, ull IDX, uint h0, uint h1, uint h2, uint h3, uint h4, uint nbits) {
	ull id = threadIdx.x | (ull)blockIdx.x << 8 | (ull)blockIdx.y << 20 | (ull)IDX << 32, idx = id;

	uint w[16];
	for(int i = 0; i < PADDING/4; i++, idx >>= 16){
		w[i] = 0x40404040u | (idx&15) << 24 | (idx>>4&15) << 16 | (idx>>8&15) << 8 | (idx>>12&15);
	}
	w[PADDING/4] = 0x0a800000;
	for(int i = PADDING/4+1; i < 15; i++) w[i] = 0;
	w[15] = nbits;

	uint a, b, c, d, e, f, k;
	a = h0; b = h1; c = h2; d = h3; e = h4;
	for(int i = 0; i < 16; i++){
		f = (b&c)|(~b&d);
		k = 0x5A827999;
		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	for(int i = 16; i < 20; i++){
		w[i&15] = LR_dev(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b&c)|(~b&d);
		k = 0x5A827999;

		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	for(int i = 20; i < 40; i++){
		w[i&15] = LR_dev(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b^c^d);
		k = 0x6ED9EBA1; 

		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	for(int i = 40; i < 60; i++){
		w[i&15] = LR_dev(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b&c)|(b&d)|(c&d);
		k = 0x8F1BBCDC;

		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	for(int i = 60; i < 80; i++){
		w[i&15] = LR_dev(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = b^c^d;
		k = 0xCA62C1D6;

		uint tmp = LR_dev(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR_dev(b, 30); b = a; a = tmp;
	}
	h0 = h0 + a;
	h1 = h1 + b;
	h2 = h2 + c;
	h3 = h3 + d;
	h4 = h4 + e;
//	if(h0 == 0 && (h1 >> 20) == 0) *res = id;
	if(h0 == 0) *res = id;
}

struct sha1{
	sha1(uint h0, uint h1, uint h2, uint h3, uint h4, uint nbits):
		h0(h0), h1(h1), h2(h2), h3(h3), h4(h4), 
		nbits(nbits){}
	uint h0, h1, h2, h3, h4, nbits;
};

string modify_message()
{
	string log = "";
	char c;

	while(scanf("%c", &c) != EOF) log += c; log.pop_back();

	while(log.size() % 64 != 0) log += " ";
	for(int t = 0; t < PADDING; t++) log += "@";
	
	log += "\n";
	
	int l = log.size()-1;
	for(int i = 0; i < log.size() && log[i]; i++) l--;
	
	log += "\x0";
	
	int m = 1, p;
	while(l >= m*10) m *= 10;
	for(p = 7; m; p++, m /= 10) log[p] = l/m + '0', l %= m;
	
	if(log[p] != '\0'){
		printf("message length's digit is changed while modifying\n");
		printf("Please add or remove +- 50? letters \n");
		exit(-1);
	}
	return log;
}

union block{
	uint v[16];
	char s[64];
};

uint LR(uint a, int x){
	return a << x | a >> 32-x;
}

sha1 get_hash(string log){
	uint h0, h1, h2, h3, h4;
	block buf[10000] = {};
	h0 = 0x67452301;
	h1 = 0xEFCDAB89;
	h2 = 0x98BADCFE;
	h3 = 0x10325476;
	h4 = 0xC3D2E1F0;

	memcpy(buf, log.c_str(), log.size());
	int len = log.size(), nbits = len * 8;
	buf[0].s[len++] = 0x80;

	uint nblock = (len+7)/64 + 1;
	buf[nblock-1].v[14] = 0;
	buf[nblock-1].v[15] = nbits;

	for(int t = 0; t+1 < nblock; t++){
		block cur = buf[t];
		for(int i = 0; i < 16; i++){
			if(t == nblock-1 && i == 15) continue;
			swap(cur.s[i*4+0], cur.s[i*4+3]);
			swap(cur.s[i*4+1], cur.s[i*4+2]);
		}
		uint w[16];
		for(int i = 0; i < 16; i++) w[i] = cur.v[i];

		uint a, b, c, d, e, f, k;
		a = h0; b = h1; c = h2; d = h3; e = h4;
		for(int i = 0; i < 80; i++){
			if(i >= 16) w[i%16] = LR(w[(i-3+16)%16]^w[(i-8+16)%16]^w[(i-14+16)%16]^w[(i-16+16)%16], 1);
			if(i <= 19){
				f = (b&c)|(~b&d);
				k = 0x5A827999;
			}
			else if(i <= 39){
				f = (b^c^d);
				k = 0x6ED9EBA1; 
			}
			else if(i <= 59){
				f = (b&c)|(b&d)|(c&d);
				k = 0x8F1BBCDC;
			}
			else if(i <= 79){
				f = b^c^d;
				k = 0xCA62C1D6;
			}
			uint tmp = LR(a, 5) + f + e + k + w[i%16];
			e = d; d = c; c = LR(b, 30); b = a; a = tmp;
		}
		h0 = h0 + a;
		h1 = h1 + b;
		h2 = h2 + c;
		h3 = h3 + d;
		h4 = h4 + e;
	}
	return sha1(h0, h1, h2, h3, h4, nbits);
}

void run_on_gpu(string log, sha1 hash){
	ull *res;
	ull res_copy;
	dim3 threadsPerBlock(256, 1);
	dim3 numBlocks(4096, 4096);

	hipMalloc(&res, sizeof(ull));

	for(int i = 0;; i++){
		sha1_kernel<<<numBlocks, threadsPerBlock>>>(res, i, hash.h0, hash.h1, hash.h2, hash.h3, hash.h4, hash.nbits);

		hipMemcpy(&res_copy, res, sizeof(ull), hipMemcpyDeviceToHost);
		if(res_copy) break;
	}
	cout << log.substr(0, (int)log.size() - PADDING-1);
	for(int i = 0; i < PADDING; i++){
		printf("%c", 64 | res_copy&15);
		res_copy /= 16;
	}
}

int main()
{
	string log = modify_message();
	sha1 hash = get_hash(log);
	run_on_gpu(log, hash);
}
