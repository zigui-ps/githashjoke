
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <getopt.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdbool.h>
#include <sys/time.h>

typedef unsigned int uint;
typedef unsigned long long ull;

__device__ uint LR(uint a, int x){
	return a << x | a >> 32-x;
}

__global__ void sha1_kernel(ull* res, ull IDX) {
	ull id = threadIdx.x | (ull)blockIdx.x << 10 | (ull)blockIdx.y << 20 | (ull)IDX << 32, idx = id;
	uint h0, h1, h2, h3, h4;
	h0 = 0x1b0a7bd1; h1 = 0x447cd35b; h2 = 0x521b5a11; h3 = 0x410602b0; h4 = 0x1896a106;

	uint w[16];
	for(int i = 0; i < 8; i++, idx >>= 16){
		w[i] = 0x40404040u | (idx&15) << 24 | (idx>>4&15) << 16 | (idx>>8&15) << 8 | (idx>>12&15);
	}
	w[8] = 0x0a800000;
	for(int i = 9; i < 15; i++) w[i] = 0;
	w[15] = 1800;

	uint a, b, c, d, e, f, k;
	a = h0; b = h1; c = h2; d = h3; e = h4;
	for(int i = 0; i < 16; i++){
		f = (b&c)|(~b&d);
		k = 0x5A827999;
		uint tmp = LR(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR(b, 30); b = a; a = tmp;
	}
	for(int i = 16; i < 20; i++){
		w[i&15] = LR(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b&c)|(~b&d);
		k = 0x5A827999;

		uint tmp = LR(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR(b, 30); b = a; a = tmp;
	}
	for(int i = 20; i < 40; i++){
		w[i&15] = LR(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b^c^d);
		k = 0x6ED9EBA1; 

		uint tmp = LR(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR(b, 30); b = a; a = tmp;
	}
	for(int i = 40; i < 60; i++){
		w[i&15] = LR(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = (b&c)|(b&d)|(c&d);
		k = 0x8F1BBCDC;

		uint tmp = LR(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR(b, 30); b = a; a = tmp;
	}
	for(int i = 60; i < 80; i++){
		w[i&15] = LR(w[i-3&15]^w[i-8&15]^w[i-14&15]^w[i&15], 1);

		f = b^c^d;
		k = 0xCA62C1D6;

		uint tmp = LR(a, 5) + f + e + k + w[i&15];
		e = d; d = c; c = LR(b, 30); b = a; a = tmp;
	}
	h0 = h0 + a;
	h1 = h1 + b;
	h2 = h2 + c;
	h3 = h3 + d;
	h4 = h4 + e;
	if(h0 == 0 && (h1 >> 16) == 0) *res = id;
}

const int SZ = 4096 * 4096;

ull *res;
ull res_copy;

int main()
{
	dim3 threadsPerBlock(1024, 1);
	dim3 numBlocks(1024, 4096);

	hipMalloc(&res, sizeof(ull));

	for(int i = 0;; i++){
		printf("IDX : %d\n", i);
		sha1_kernel<<<numBlocks, threadsPerBlock>>>(res, i);

		hipMemcpy(&res_copy, res, sizeof(ull), hipMemcpyDeviceToHost);
		if(res_copy == 0) continue;
		for(int i = 0; i < 16; i++){
			printf("%c", 64 | res_copy&15);
			res_copy /= 16;
		}
		break;
	}
}
